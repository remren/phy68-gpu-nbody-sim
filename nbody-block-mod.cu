#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include "timer.h"

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float4 *pos, *vel; } BodySystem;

void save_positions(const float4* positions, int nBodies, int timestep, FILE* file) {
    fwrite(&timestep, sizeof(int), 1, file);
    for (int i = 0; i < nBodies; i++) {
        fwrite(&positions[i].x, sizeof(float), 3, file); // Save only x,y,z
    }
}

void randomizeBodies(float *data, int n) {
    for (int i = 0; i < n; i += 4) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        data[i + 1] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        data[i + 2] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
        data[i + 3] = 1.0f;
    }
    // Make the 0 element mass larger than others.
    data[3] = 10000000.0f;
}

__global__
void bodyForce(float4 *p, float4 *v, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

        for (int tile = 0; tile < gridDim.x; tile++) {
            __shared__ float3 spos[BLOCK_SIZE];
            float4 tpos = p[tile * blockDim.x + threadIdx.x];
            spos[threadIdx.x] = make_float3(tpos.x, tpos.y, tpos.z);
            __syncthreads();

            for (int j = 0; j < BLOCK_SIZE; j++) {
                float dx = spos[j].x - p[i].x;
                float dy = spos[j].y - p[i].y;
                float dz = spos[j].z - p[i].z;
                float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
                float invDist = rsqrtf(distSqr);
                float invDist3 = invDist * invDist * invDist;

                Fx += dx * invDist3 * tpos.w;
                Fy += dy * invDist3 * tpos.w;
                Fz += dz * invDist3 * tpos.w;
            }
            __syncthreads();
        }

        v[i].x += dt*Fx; v[i].y += dt*Fy; v[i].z += dt*Fz;
    }
}

int main(const int argc, const char** argv) {
    int nBodies = 100;
    if (argc > 1) nBodies = atoi(argv[1]);

    const float dt = 0.01f;
    const int nIters = 1000;

    int bytes = 2*nBodies*sizeof(float4);
    float *buf = (float*)malloc(bytes);
    BodySystem p = { (float4*)buf, ((float4*)buf) + nBodies };

    randomizeBodies(buf, 8*nBodies);

    float *d_buf;
    hipMalloc(&d_buf, bytes);
    BodySystem d_p = { (float4*)d_buf, ((float4*)d_buf) + nBodies };

    int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
    double totalTime = 0.0;

    // Open output file
    FILE* output_file = fopen("large_mass_particle_positions.bin", "wb");
    if (!output_file) {
        printf("Error opening output file!\n");
        return 1;
    }

    // Write header (nBodies, nIters, nMasses)
    fwrite(&nBodies, sizeof(int), 1, output_file);
    fwrite(&nIters, sizeof(int), 1, output_file);
    for (int i = 0; i < 8*nBodies; i += 4) {
        fwrite(&buf[i + 3], sizeof(float), 1, output_file); // save all masses (+ 3 is w struct field), same logic as randomizeBodies
    }

    for (int iter = 1; iter <= nIters; iter++) {
        StartTimer();

        hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
        bodyForce<<<nBlocks, BLOCK_SIZE>>>(d_p.pos, d_p.vel, dt, nBodies);
        hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

        for (int i = 0; i < nBodies; i++) {
            p.pos[i].x += p.vel[i].x*dt;
            p.pos[i].y += p.vel[i].y*dt;
            p.pos[i].z += p.vel[i].z*dt;
        }

        // Save positions for this timestep
        save_positions(p.pos, nBodies, iter, output_file);

        const double tElapsed = GetTimer() / 1000.0;
        if (iter > 1) {
            totalTime += tElapsed;
        }
#ifndef SHMOO
        printf("Iteration %d: %.3f seconds\n", iter, tElapsed);
#endif
    }

    fclose(output_file);
    double avgTime = totalTime / (double)(nIters-1);

#ifdef SHMOO
    printf("%d Bodies, %d Iterations, average of %0.3f Billion Interactions / sec, totalTime=%f\n",
           nBodies, nIters, 1e-9 * nBodies * nBodies / avgTime, totalTime);
#else
    printf("Average rate: %.3f Billion Interactions / second\n",
           1e-9 * nBodies * nBodies / avgTime);
#endif

    free(buf);
    hipFree(d_buf);
    return 0;
}